#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "kernel.h"
#include "kernel.cu"
#include "array.h"
#include <math.h>

using namespace std;

int main()
{
    int N = 4096;
    int SIZE = N*N;

    vector<float> h_A(SIZE);
    vector<float> h_B(SIZE);
    vector<float> h_C(SIZE);

    for (int i=0; i<N; i++){
        for (int j=0; j<N; j++){
            h_A[i*N+j] = sin(i);
            h_B[i*N+j] = cos(j);
        }
    }

    array<float> d_A(SIZE);
    array<float> d_B(SIZE);
    array<float> d_C(SIZE);

    d_A.set(&h_A[0], SIZE);
    d_B.set(&h_B[0], SIZE);

    matrixMultiplication(d_A.getData(), d_B.getData(), d_C.getData(), N);
    hipDeviceSynchronize();

    d_C.get(&h_C[0], SIZE);
    hipDeviceSynchronize();

    return 0;
}